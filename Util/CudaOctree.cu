#include "hip/hip_runtime.h"
/**
 *  CPE 2013
 *  -------------------
 *  Program
 *
 *  Last Modified:
 *  @author Nick Feeney
 */
#define CUDASAFECALL( call )  CUDA_SAFE_CALL( call )
#include "cutil.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <math.h>
#include "UtilTypes.h"
#include "Octree.h"
#define CUDAERRORCHECK() {                   \
   hipError_t err = hipGetLastError();        \
   if( hipSuccess != err){ \
      printf("CudaErrorCheck %d\n", err);           \
      exit(1); \
   } }
#define PI 3.14159265359
#define MONTE_CARLO_N 256
#define MAX_DEPTH  20
#define MAX_OCTREE_SIZE 1000

__device__ Hermonics gpuCalculateSphericalHermonics( struct Surfel surfel );
__device__ Hermonics gpuCreateHermonics();
__device__ void gpuAddHermonics( Hermonics &save, Hermonics &gone );
void cpuAddHermonics( Hermonics &save, Hermonics &gone );
__global__ void fillLeafSphericalHermonics( CudaNode *d_root, int tree_total,
      Surfel *surfels, int surfel_total, int *d_leaf_addrs, int leaf_nodes );
__global__ void kernel_FirstPassSphericalHermonics( Surfel *d_surfels, Hermonics *d_hermonics,
      int num, int batch, int batch_size );
__global__ void kernel_SecondPassSphericalHermonics( CudaNode *d_root, int nodes,
      Hermonics *d_hermonics, int num_her, int *d_leaf_addrs, int leafs );
extern "C" int getTime( );
extern "C" float getDiffTime( int start, int end );
void FillOutHermonicsFromArray( int current, CudaNode *root, Hermonics *hermonics );
__device__ vec3 gpuUnit( vec3 in )
{
   float mag = sqrt(in.x*in.x + in.y *in.y + in.z*in.z);
   in.x /= mag;
   in.y /= mag;
   in.z /= mag;
   return in;
}
void checkCUDAError(const char *msg) {
   hipError_t err = hipGetLastError();
   if( hipSuccess != err) {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
      exit(EXIT_FAILURE);
   }
}
__device__ Hermonics gpuCalculateSphericalHermonics( struct Surfel surfel, int seed )
{
   double red[9];
   double green[9];
   double blue[9];
   double areas[9];
   for( int i = 0; i < 9; i++ )
   {
      red[i] = 0;
      green[i] = 0;
      blue[i] = 0;
      areas[i] = 0;
   }

   double area = PI * surfel.radius * surfel.radius;
   Color inColor;
   inColor.r = fmin(surfel.color.r, (float)0.999);
   inColor.g = fmin(surfel.color.g, (float)0.999);
   inColor.b = fmin(surfel.color.b, (float)0.999);
   surfel.normal = gpuUnit(surfel.normal);
   hiprandState s;
   hiprand_init(seed, 0, 0, &s);

   //Weighted Stocasically sample phi from 0 to 2pi

   double TYlm[9];
   double phi;
   double theta;

   double sin_theta;
   double cos_theta;
   double sin_phi;
   double cos_phi;
   double dx;
   double dy;
   double dz;
   double tx,ty,tz;

   double d_dot_n;
   double x,y;
   for( int j = 0; j < MONTE_CARLO_N; j++ )
   {
      for( int i = 0; i < MONTE_CARLO_N; i++ )
      {
         x = ((double)j + hiprand_uniform_double(&s) ) / MONTE_CARLO_N;
         y = ((double)i + hiprand_uniform_double(&s)) / MONTE_CARLO_N;

         phi = 2.0 * PI * y;
         theta = 2.0 * acos( sqrt( 1.0 - x ) );

         sin_theta = sin(theta);
         cos_theta = cos(theta);
         sin_phi = sin(phi);
         cos_phi = cos(phi);
         dx = sin_theta*cos_phi;
         dy = sin_theta*sin_phi;
         dz = cos_theta;

         d_dot_n = dx * (double)surfel.normal.x;
         d_dot_n += dy * (double)surfel.normal.y;
         d_dot_n += dz * (double)surfel.normal.z;

         tx = sin_theta * cos_phi;
         ty = sin_theta * sin_phi;
         tz = cos_theta;
         TYlm[0] = 0.282095; //0 0
         TYlm[1] = .488603 * -ty;//1 -1
         TYlm[2] = .488603 * tz;//1 0
         TYlm[3] = .488603 * -tx; //1 1
         TYlm[4] = 1.092548 * tx * ty; // 2 -2
         TYlm[5] = 1.092548 * -ty * tz; //2 -1
         TYlm[6] = 0.315392 * (3*tz*tz - 1); //2 0
         TYlm[7] = 1.092548 * -tx * tz; //2 1
         TYlm[8] = .546274 * (tx*tx - ty*ty); //2 2

         //now > 0
         if(d_dot_n > 0.001)
         {
            for( int k = 0; k < 9; k++ )
            {
               double a = (area * d_dot_n * TYlm[k] * sin_theta);
               /*
               //Red
               red[k] += (double)inColor.r * a;
               //Green
               green[k] += (double)inColor.g * a;
               //Blue
               blue[k] += (double)inColor.b * a;
               //area
               */
               areas[k] += a;
            }
         }
      }
   }
   for( int k = 0; k < 9; k++ )
   {
      red[k] = areas[k] * inColor.r;
      green[k] = areas[k] * inColor.g;
      blue[k] = areas[k] * inColor.b;
   }
   double factor = ((4.0*PI)/((double)MONTE_CARLO_N*(double)MONTE_CARLO_N));
   for( int j = 0; j < 9; j++ )
   {
      red[j] *= factor;
      green[j] *= factor;
      blue[j] *= factor;
      areas[j] *= factor;
   }
   Hermonics sh;
   for( int i =0; i < 9; i++ )
   {
      sh.red[i] = red[i];
      sh.green[i] = green[i];
      sh.blue[i] = blue[i];
      sh.area[i] = areas[i];
   }
   return sh;
}
__device__ Hermonics gpuCreateHermonics()
{
   Hermonics sh;
   for( int i =0; i< 9; i++ )
   {
      sh.red[i] = 0;
      sh.green[i] = 0;
      sh.blue[i] = 0;
      sh.area[i] = 0;
   }
   return sh;
}
void cpuAddHermonics( Hermonics &save, Hermonics &gone )
{
   for( int j= 0; j < 9; j++ )
   {
      save.red[j] += gone.red[j];
      save.green[j] += gone.green[j];
      save.blue[j] += gone.blue[j];
      save.area[j] += gone.area[j];
   }
}
__device__ void gpuAddHermonics( Hermonics &save, Hermonics &gone )
{
   for( int j= 0; j < 9; j++ )
   {
      save.red[j] += gone.red[j];
      save.green[j] += gone.green[j];
      save.blue[j] += gone.blue[j];
      save.area[j] += gone.area[j];
   }
}
__global__ void testGPU()
{
   printf("test\n");
}
extern "C" void testME( )
{
   printf("This\n");
   testGPU<<<1,2>>>();
   hipDeviceSynchronize();
   CUDAERRORCHECK();
   printf("Happened\n");
}
__global__ void fillLeafSphericalHermonics( CudaNode *d_root, int tree_total,
      Surfel *surfels, int surfel_total, int *d_leaf_addrs, int leaf_nodes,
      int batch, int batch_size )
{
   int leaf_addr_index = blockIdx.x + batch * batch_size;
   int surfel_index = threadIdx.x;

   if( leaf_addr_index >= leaf_nodes)
      return;
   if( surfel_index > 32 || surfel_index < 0 )
      return;
   leaf_addr_index = 0;
   surfel_index = 0;
   __shared__ int leaf_node_index;
   __shared__ int surfel_start;
   __shared__ int surfel_end;
   leaf_node_index = d_leaf_addrs[leaf_addr_index];
   surfel_start = d_root[leaf_node_index].children[0];
   surfel_end = d_root[leaf_node_index].children[1];
   leaf_node_index = 0 ;
   surfel_start = 0;
   surfel_end = 20;

   if( leaf_node_index >= tree_total || surfel_start < 0 || surfel_end < 0 || surfel_start >= surfel_total || surfel_end >= surfel_total )
      return;
   if( surfel_start == surfel_end )
   {
      if( threadIdx.x == 0 )
         d_root[leaf_node_index].hermonics = gpuCreateHermonics();
      return;
   }
   __shared__ Hermonics hermonics[1];

   if( surfel_start + surfel_index < surfel_end )
      hermonics[surfel_index] = gpuCalculateSphericalHermonics(surfels[surfel_start + surfel_index],
            leaf_addr_index * surfel_index);

   return;
   __syncthreads();

   if( threadIdx.x == 0 )
   {
      for( int i = 1; i < surfel_end - surfel_start; i++ )
         gpuAddHermonics( hermonics[0], hermonics[i] );
      d_root[leaf_node_index].hermonics = hermonics[0];
   }
}
__global__ void kernel_FirstPassSphericalHermonics( Surfel *d_surfels, Hermonics *d_hermonics,
      int num, int batch, int batch_size )
{
   int index = (blockIdx.x + batch*batch_size) * 32 + threadIdx.x;

   if(index > num || threadIdx.x >= 32)
      return;

   Surfel s = d_surfels[index];
   Hermonics temp = gpuCalculateSphericalHermonics( s, index );

   d_hermonics[index] = temp;
}
extern "C" void gpuTestFirstPassSphericalHermonics( CudaNode *root, int nodes, SurfelArray &SA,
      int *leaf_addrs, int leaf_nodes )
{
   printf("leaf_addrs: %d, Surfels: %d, CudaNodes: %d\n", leaf_nodes, SA.num, nodes );
   float surfel_size = (float)(sizeof(Surfel) * SA.num)/1048576.0;
   float hermonics_size = (float)(sizeof(Hermonics) * SA.num)/1048576.0;
   printf("Sizes: Surfel %f Hermonics %f\n Total %f\n", surfel_size, hermonics_size,
         surfel_size + hermonics_size);
   CudaNode *d_root;
   Surfel *d_surfels;
   Hermonics *d_hermonics;
   Hermonics *hermonics = (Hermonics *) malloc( sizeof(Hermonics) * SA.num );
   int * d_leaf_addrs;
   int num_blocks = ceilf((float)SA.num / 32.0);
   int batch_size = 50;
   int batches = ceilf( (float)num_blocks/batch_size );

   dim3 dimBlock( 32 );
   dim3 dimGrid( batch_size );

   CUDASAFECALL(hipMalloc( (void **)&d_surfels, sizeof(Surfel) * SA.num));
   CUDASAFECALL(hipMalloc( (void **)&d_hermonics, sizeof(Hermonics) * SA.num));

   CUDASAFECALL(hipMemcpy( d_surfels, SA.array, sizeof(Surfel) * SA.num,
            hipMemcpyHostToDevice ));

   hipEvent_t t0,t1;
   hipEventCreate(&t0);
   hipEventCreate(&t1);
   hipEventRecord( t0, 0 );
   hipDeviceSynchronize();
   for( int i = 0; i < batches; i++ )
   {
      kernel_FirstPassSphericalHermonics<<<dimGrid, dimBlock>>>( d_surfels, d_hermonics, SA.num,
            i, batch_size );
   }

   CUDAERRORCHECK();
   hipEventRecord( t1, 0 );
   hipEventSynchronize( t1 );
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime, t0, t1);
   printf("Time for First Pass: %f\n", elapsedTime/1000 );
   hipEventDestroy( t1 );
   hipEventDestroy( t0 );

   CUDASAFECALL(hipMemcpy( hermonics, d_hermonics, sizeof(Hermonics) * SA.num,
            hipMemcpyDeviceToHost ));
   CUDASAFECALL(hipFree( d_surfels));
   CUDASAFECALL(hipFree( d_hermonics));
   hipDeviceSynchronize();

   printf("Starting CPU FILL\n");
   FillOutHermonicsFromArray( 0, root, hermonics );
   free(hermonics);
   printf("Ending\n");
}
extern "C" void gpuTwoPassSphericalHermonics( CudaNode *root, int nodes, SurfelArray &SA,
      int *leaf_addrs, int leaf_nodes )
{
   printf("Leaf_nodes %d", leaf_nodes );
   CudaNode *d_root;
   Surfel *d_surfels;
   Hermonics *d_hermonics;
   int * d_leaf_addrs;
   int num_blocks = ceilf((float)SA.num / 32.0);
   int batch_size = 50;
   int batches = ceilf( (float)num_blocks/batch_size );

   dim3 dimBlock( 32 );
   dim3 dimGrid( batch_size );

   CUDASAFECALL(hipMalloc( (void **)&d_surfels, sizeof(Surfel) * SA.num));
   CUDASAFECALL(hipMalloc( (void **)&d_hermonics, sizeof(Hermonics) * SA.num));

   CUDASAFECALL(hipMemcpy( d_surfels, SA.array, sizeof(Surfel) * SA.num,
            hipMemcpyHostToDevice ));

   for( int i = 0; i < batches; i++ )
   {
      kernel_FirstPassSphericalHermonics<<<dimGrid, dimBlock>>>( d_surfels, d_hermonics, SA.num,
            i, batch_size );
   }

   CUDASAFECALL(hipFree( d_surfels));

   CUDASAFECALL(hipMalloc( (void **)&d_root, sizeof(CudaNode) * nodes));
   CUDASAFECALL(hipMalloc( (void **)&d_leaf_addrs, sizeof(int) * leaf_nodes));

   CUDASAFECALL(hipMemcpy( d_root, root, sizeof(CudaNode) * nodes, hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpy( d_leaf_addrs, leaf_addrs, sizeof(int) * leaf_nodes,
            hipMemcpyHostToDevice ));

   num_blocks = ceilf((float)leaf_nodes / 32.0);
   dim3 dimGrid2( num_blocks );
   kernel_SecondPassSphericalHermonics<<<dimGrid, dimBlock>>>( d_root, nodes, d_hermonics, SA.num,
         d_leaf_addrs, leaf_nodes );


   CUDASAFECALL(hipMemcpy( root, d_root, sizeof(CudaNode) * nodes, hipMemcpyDeviceToHost ));
   hipFree( d_root );
   hipFree( d_hermonics );
   hipFree( d_leaf_addrs );
}
__global__ void  kernel_SecondPassSphericalHermonics( CudaNode *d_root, int nodes,
      Hermonics *d_hermonics, int num, int *d_leaf_addrs, int leafs )
{
}
extern "C" void gpuFilloutSphericalHermonics( CudaNode *root, int nodes, SurfelArray &SA,
      int *leaf_addrs, int leaf_nodes )
{
   int batch_size = 5;
   int batches = (leaf_nodes + batch_size - 1)/batch_size;
   dim3 dimBlock( 32 );
   dim3 dimGrid( batch_size );

   CudaNode *d_root;
   Surfel *d_surfels;
   int * d_leaf_addrs;

   printf("Leaf_nodes: %d\n", leaf_nodes );
   fflush(stdout);

   hipEvent_t t0,t1,t2,t3;
   hipEventCreate(&t0);
   hipEventCreate(&t1);
   hipEventCreate(&t2);
   hipEventCreate(&t3);
   hipEventRecord( t0, 0 );

   CUDASAFECALL(hipMalloc( (void **)&d_root, sizeof(CudaNode) * nodes));
   CUDASAFECALL(hipMalloc( (void **)&d_surfels, sizeof(Surfel) * SA.num));
   CUDASAFECALL(hipMalloc( (void **)&d_leaf_addrs, sizeof(int) * leaf_nodes));

   CUDASAFECALL(hipMemcpy( d_root, root, sizeof(CudaNode) * nodes, hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpy( d_surfels, SA.array, sizeof(Surfel) * SA.num,
            hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpy( d_leaf_addrs, leaf_addrs, sizeof(int) * leaf_nodes,
            hipMemcpyHostToDevice ));

   hipDeviceSynchronize();

   hipEventRecord( t1, 0 );
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime, t0, t1);
   printf("time %f\n", elapsedTime );

   CUDAERRORCHECK();
   for( int i = 0; i < batches; i++ )
   {
      fillLeafSphericalHermonics<<<dimGrid, dimBlock>>>( d_root, nodes, d_surfels, SA.num,
            d_leaf_addrs, leaf_nodes, 0, batch_size );
      hipDeviceSynchronize();
      CUDAERRORCHECK();
      fprintf(stderr, "%d/%d\n", i, batches );
   }
   CUDAERRORCHECK();
   hipEventRecord( t2, 0 );
   float elapsedTime2;
   hipEventElapsedTime(&elapsedTime2, t1, t2);
   printf("Batched Time %f\n", elapsedTime );


   CUDASAFECALL(hipMemcpy( root, d_root, sizeof(CudaNode) * nodes, hipMemcpyDeviceToHost ));
   hipFree( d_root );
   hipFree( d_surfels );
   hipFree( d_leaf_addrs );
}
hipEvent_t times[100];
int current = 0;
extern "C" int getTime( )
{
   if( current >= 100 )
      return -1;
   hipEventCreate(times+current);
   hipEventRecord( times[current], 0 );
   current++;
   return current-1;
}
extern "C" float getDiffTime( int start, int end )
{
   if(start < 0 || end < 0)
      return -1;
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime, times[start], times[end]);
   return elapsedTime;
}
/*
   extern "C" Hermonics gpuCalculateSphericalHermonics( Surfel &surfel, int sqrt_samples )
   {
   dim3 dimBlock( sqrt_samples );
   dim3 dimGrid( leaf_nodes );

   Hermonics *d_hermonics;
   CUDASAFECALL(hipMalloc( (void **)&d_hermonics, sizeof(Hermonics)));

   gpu_kernel_calc_spherical_hermonics( surfel, sqrt_samples, d_hermoncis );
   Hermonics ret;

   CUDASAFECALL(hipMemcpy( ret, d_hermonics, sizeof(Hermonics), hipMemcpyDeviceToHost ));
   }
 */
void FillOutHermonicsFromArray( int current, CudaNode *root, Hermonics *hermonics )
{
   root[current].hermonics = createHermonics();
   if( root[current].leaf )
   {
      for( int i = root[current].children[0]; i < root[current].children[1]; i++ )
         cpuAddHermonics(root[current].hermonics, hermonics[i]);
      return;
   }
   for( int i = 0; i < 8; i++ )
   {
      FillOutHermonicsFromArray(root[current].children[i], root, hermonics );
      cpuAddHermonics(root[current].hermonics, root[root[current].children[i]].hermonics);
   }

}
