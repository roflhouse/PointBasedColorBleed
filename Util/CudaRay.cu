#include "hip/hip_runtime.h"
/**
 *  CPE 2012
 *  -------------------
 *  Program
 *
 *  Last Modified:
 *  @author Nick Feeney
 */

#include <stdio.h>
#include "cutil.h"
#include "Octree.h"
#define MAX_OCTREE_DEPTH 40
#define MAX_ANGLE 0.00
#define RADIUS 0.01
#include "UtilTypes.h"
#include "RasterCube.h"
#include "../Objects/SurfelType.h"
#define CUDASAFECALL( call )  CUDA_SAFE_CALL( call )
#define CUDAERRORCHECK() {                   \
   hipError_t err = hipGetLastError();        \
   if( hipSuccess != err){ \
      printf("CudaErrorCheck %d\n", err);           \
      exit(1); \
   } }


__device__ float gpuDot(const vec3 &one, const vec3 &two);
__device__ float surfelHitTestCuda( Surfel s, Ray &ray );
__device__ float squareDistanceCuda( vec3 &one, vec3 &two );
__device__ vec3 gpuUnit(vec3 &in);
__device__ float magCuda(const vec3 &in);
__device__ bool testForHitCuda( BoundingBox &boxIn, Ray &ray );
__device__ Surfel gpu_raytrace( CudaNode *gpu_root, Surfel *gpu_array, Ray &ray );
__device__ bool gpuBBInTest( const BoundingBox &box, const vec3 &pos );

__global__ void kernel_CastRays( CudaNode *gpu_root, Surfel *gpu_array,
      int surfels, Ray *gpu_rays, int num_rays, Surfel *output );

extern "C" Surfel *gpuCastRays( CudaNode *cpu_root, int nodes, SurfelArray cpu_array,
      Ray *rays, int num_rays )
{
   printf("Surfels: %d, CudaNodes: %d, Rays: %d\n", cpu_array.num, nodes, num_rays );
   float surfel_size = (float)(sizeof(Surfel) * cpu_array.num)/1048576.0;
   float cn_size = (float)(sizeof(CudaNode) * nodes)/1048576.0;
   float ray_size = (float)(sizeof(Ray) * num_rays)/1048576.0;
   float output_size = (float)(sizeof(Surfel) * num_rays )/1048576.0;
   printf("Sizes: Surfel %f CudaNodes %f Rays %f output: %f\n Total %f\n",
         surfel_size, cn_size, ray_size, output_size,
         surfel_size + cn_size + ray_size, output_size);

   CudaNode * d_root;
   Surfel *d_surfels;
   Ray *d_rays;
   Surfel *d_output;
   Surfel *cpu_output = (Surfel *)malloc( sizeof(Surfel)*num_rays );

   CUDASAFECALL(hipMalloc( (void **)&d_surfels, sizeof(Surfel) * cpu_array.num));
   CUDASAFECALL(hipMalloc( (void **)&d_root, sizeof(CudaNode) * nodes));
   CUDASAFECALL(hipMalloc( (void **)&d_rays, sizeof(Ray) * num_rays));
   CUDASAFECALL(hipMalloc( (void **)&d_output, sizeof(Surfel) * num_rays ));

   CUDASAFECALL(hipMemcpy( d_surfels, cpu_array.array, sizeof(Surfel) * cpu_array.num,
            hipMemcpyHostToDevice));
   CUDASAFECALL(hipMemcpy( d_root, cpu_root, sizeof(CudaNode) * nodes,hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpy( d_rays, rays, sizeof(Ray) * num_rays,hipMemcpyHostToDevice ));

   int num_blocks = ceilf( (float)num_rays / 32.0 );
   dim3 dimBlock( 32 );
   dim3 dimGrid( num_blocks );

   printf("GPU Casting Rays\n");
   kernel_CastRays<<<dimGrid, dimBlock>>>( d_root, d_surfels, cpu_array.num, d_rays,
         num_rays, d_output );
   printf("Done GPU casting\n");
   CUDAERRORCHECK();

   CUDASAFECALL(hipMemcpy( cpu_output, d_output, sizeof(Surfel) * num_rays,
            hipMemcpyDeviceToHost ));
   hipFree( d_output );
   hipFree( d_surfels );
   hipFree( d_root );
   hipFree( d_rays );

   return cpu_output;
}

__global__ void kernel_CastRays( CudaNode *gpu_root, Surfel *gpu_array,
      int surfels, Ray *gpu_rays, int num_rays, Surfel *output )
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;

   if( index >= num_rays )
      return;
   Ray ray = gpu_rays[index];

   Surfel ret = gpu_raytrace( gpu_root, gpu_array, ray );

   output[index] = ret;
}
__device__ Surfel gpu_raytrace( CudaNode *gpu_root, Surfel *gpu_array, Ray &ray )
{
   int stack[MAX_OCTREE_DEPTH*8+2];
   bool hit = false;
   float t = 0;
   float bestT = 100000;
   Surfel bestSurfel;
   int stack_current = 1;
   CudaNode cached;

   //push root on stack;
   stack[0] = 0;
   while( stack_current )
   {
      stack_current--;

      cached = gpu_root[stack[stack_current]];
      if( testForHitCuda( cached.box, ray ) )
      {
         if( cached.leaf )
         {
            for( int i = cached.children[0]; i < cached.children[1]; i++ )
            {
               t = surfelHitTestCuda( gpu_array[i], ray );
               if( (t > 0 && t < bestT) || (hit == false && t > 0) )
               {
                  bestT = t;
                  bestSurfel = gpu_array[i];
                  hit = true;
               }
            }
         }
         else
         {
            for( int i = 0; i < 8; i++ )
            {
               if( cached.children[i] > 0 ){
                  stack[stack_current] = cached.children[i];
                  stack_current++;
               }
            }
         }
      }
   }
   if( hit )
   {
      vec3 hitMark;
      hitMark.x = ray.dir.x * bestT + ray.pos.x;
      hitMark.y = ray.dir.y * bestT + ray.pos.y;
      hitMark.z = ray.dir.z * bestT + ray.pos.z;
      bestSurfel.pos = hitMark;
      bestSurfel.radius = 1;
   }
   else
   {
      bestSurfel.radius = -1;
   }
   return bestSurfel;
}
__device__ float surfelHitTestCuda( Surfel s, Ray &ray )
{
   vec3 direction = gpuUnit(ray.dir);
   vec3 position;
   vec3 normal = gpuUnit(s.normal);

   position.x = ray.pos.x;
   position.y = ray.pos.y;
   position.z = ray.pos.z;

   float vd = gpuDot(normal, direction);
   if( vd > 0.001)
      return -1;
   float v0 = -(gpuDot(position, normal) + s.distance );
   float t = v0/vd;
   if( t < 0.01)
      return -1;

   vec3 hitMark;
   hitMark.x = ray.pos.x + direction.x*t;
   hitMark.y = ray.pos.y + direction.y*t;
   hitMark.z = ray.pos.z + direction.z*t;
   float d = squareDistanceCuda( hitMark, s.pos );

   if( d < s.radius*s.radius )
      return t;
   else
      return -1;
}
__device__ float squareDistanceCuda( vec3 &one, vec3 &two )
{
   return ((one.x-two.x)*(one.x-two.x) + (one.y-two.y)*(one.y-two.y) + (one.z-two.z)*(one.z-two.z));
}
__device__ vec3 gpuUnit(vec3 &in)
{
   float temp;
   vec3 newVector;
   newVector.x = 0;
   newVector.y = 0;
   newVector.z = 0;
   temp = magCuda(in);

   if(temp > 0)
   {
      newVector.x = in.x/temp;
      newVector.y = in.y/temp;
      newVector.z = in.z/temp;
   }
   return newVector;
}
__device__ float magCuda(const vec3 &in)
{
   return sqrt(in.x*in.x + in.y*in.y + in.z*in.z);
}
__device__ bool testForHitCuda( BoundingBox &boxIn, Ray &ray )
{
   vec3 min = boxIn.min;
   min.x -= RADIUS;
   min.y -= RADIUS;
   min.z -= RADIUS;

   vec3 max = boxIn.max;
   max.x += RADIUS;
   max.y += RADIUS;
   max.z += RADIUS;
   BoundingBox box;
   box.min = min;
   box.max = max;

   if( ray.dir.x > -0.0001 && ray.dir.x < 0.0001 )
   {
      if( ray.pos.x < box.min.x || ray.pos.x > box.max.x )
         return false;
   }
   if( ray.dir.y > -0.0001 && ray.dir.y < 0.0001 )
   {
      if( ray.pos.y < box.min.y || ray.pos.y > box.max.y )
         return false;
   }
   if( ray.dir.z > -0.0001 && ray.dir.z < 0.0001 )
   {
      if( ray.pos.z < box.min.z || ray.pos.z > box.max.z )
         return false;
   }
   float txmin = (box.min.x - ray.pos.x) / ray.dir.x;
   float tymin = (box.min.y - ray.pos.y) / ray.dir.y;
   float tzmin = (box.min.z - ray.pos.z) / ray.dir.z;
   float txmax = (box.max.x - ray.pos.x) / ray.dir.x;
   float tymax = (box.max.y - ray.pos.y) / ray.dir.y;
   float tzmax = (box.max.z - ray.pos.z) / ray.dir.z;

   if( txmin > txmax )
   {
      float temp = txmax;
      txmax = txmin;
      txmin = temp;
   }
   if( tymin > tymax )
   {
      float temp = tymax;
      tymax = tymin;
      tymin = temp;
   }
   if( tzmin > tzmax )
   {
      float temp = tzmax;
      tzmax = tzmin;
      tzmin = temp;
   }

   float tgmin = txmin;
   float tgmax = txmax;
   //find largest min
   if( tgmin < tymin )
      tgmin = tymin;
   if( tgmin < tzmin )
      tgmin = tzmin;

   //find smallest max
   if( tgmax > tymax )
      tgmax = tymax;
   if( tgmax > tzmax )
      tgmax = tzmax;

   if( tgmin > tgmax )
      return false;
   return true;
}
__device__ float gpuDot(const vec3 &one, const vec3 &two)
{
   return one.x*two.x + one.y*two.y + one.z*two.z;
}

__device__ float gpuDistance(const vec3 &one, const vec3 &two )
{
   return sqrt((one.x-two.x)*(one.x-two.x) + (one.y-two.y)*(one.y-two.y) +
         (one.z-two.z)*(one.z-two.z));
}
__device__ int gpuBelowHorizon( const BoundingBox &box, vec3 &position, vec3 &normal )
{
   vec3 points[8];
   points[0] = box.min;
   points[1] = box.min;
   points[1].z = box.max.z;
   points[2] = box.min;
   points[2].y = box.max.y;
   points[3] = box.min;
   points[3].y = box.max.y;
   points[3].z = box.max.z;
   points[4] = box.min;
   points[4].x = box.max.x;
   points[5] = box.min;
   points[5].x = box.max.x;
   points[5].z = box.max.z;
   points[6] = box.min;
   points[6].x = box.max.x;
   points[6].y = box.max.y;
   points[7] = box.max;
   int below = 0;
   for( int i = 0; i < 8; i++ )
   {
      vec3 temp;
      temp.x = points[i].x - position.x;
      temp.y = points[i].y - position.y;
      temp.z = points[i].z - position.z;
      temp = gpuUnit( temp );
      if( gpuDot( normal, temp ) <= 0.01 )
         below++;
   }
   return below;
}
__device__ vec3 gpuGetCenter( const BoundingBox &box )
{
   vec3 c;
   c.x = (box.max.x -box.min.x)/2 + box.min.x;
   c.y = (box.max.y -box.min.y)/2 + box.min.y;
   c.x = (box.max.z -box.min.z)/2 + box.min.z;
   return c;
}
__device__ bool gpuBBInTest( const BoundingBox &box, const vec3 &pos )
{
   if (pos.x >= box.max.x || pos.x < box.min.x )
      return false;
   if (pos.y >= box.max.y || pos.y < box.min.y )
      return false;
   if (pos.z >= box.max.z || pos.z < box.min.z )
      return false;
   return true;
}
__device__ void gpuTraverseOctreeStack( RasterCube &cube, CudaNode *gpu_root, Surfel *gpu_array,
      vec3 &position, vec3 normal, vec3 ***cuberays, glm::mat4 *cubetransforms )
{
   float dis = 0;

   int stack[MAX_OCTREE_DEPTH * 8];
   int pointer = 0;
   stack[pointer] = 0;
   pointer++;

   while( pointer )
   {
      pointer--;
      int current = stack[pointer];

      CudaNode node = gpu_root[current];
      if( node.leaf )
      {
         for( int i = node.children[0]; i < node.children[1]; i++ )
         {
            Surfel s = gpu_array[i];
            dis = gpuDistance( position, s.pos );
            if ( dis < s.radius)
            {
               gpuRaytraceSurfelToCube( cube, s, cuberays, position, normal );
            }
            else
            {
               gpuRasterizeSurfelToCube( cube, s, cubetransforms, cuberays,
                     position, normal );
            }
         }
      }
      else
      {
         if( gpuBBInTest( node.box, position ) )
         {
            for(int i = 7; i <= 0; i-- )
            {
               stack[pointer] = node.children[i];
               pointer++;
            }
            continue;
         }
         int horizon = gpuBelowHorizon( node.box, position, normal );
         if( horizon == 8 )
            continue;

         vec3 center = gpuGetCenter(node.box);

         vec3 centerToEye;
         centerToEye.x = position.x - center.x;
         centerToEye.y = position.y - center.y;
         centerToEye.z = position.z - center.z;
         centerToEye = gpuUnit(centerToEye);

         dis = distance( position, center );
         float area = gpuEvaluateSphericalHermonicsArea( node, centerToEye );
         float solidangle = area / (dis * dis);
         if( solidangle < MAX_ANGLE )
         {
            Color c = gpuEvaluateSphericalHermonicsPower( node, centerToEye );
            gpuRasterizeClusterToCube( cube, c, area, center, cubetransforms, cuberays,
                  position, normal, dis );
            continue;
         }
         else
         {
            for(int i = 7; i <= 0; i-- )
            {
               stack[pointer] = node.children[i];
               pointer++;
            }
            continue;
         }
      }
   }
}
